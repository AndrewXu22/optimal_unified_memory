#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Created by Pawan Harish.
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <adapter.h>



#define MAX_THREADS_PER_BLOCK 512

int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

#include "kernel.cu"
#include "kernel2.cu"

void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	no_of_nodes=0;
	edge_list_size=0;
	BFSGraph( argc, argv);
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}
////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph( int argc, char** argv) 
{

    char *input_f;
	if(argc!=2){
	Usage(argc, argv);
	exit(0);
	}
	
	input_f = argv[1];
	printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	int source = 0;

	fscanf(fp,"%d",&no_of_nodes);

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	// allocate host memory
                printf("alloc h_graph_nodes\n");
#if adv1==0
	Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
#else
	Node* h_graph_nodes = (Node*) xplacer_malloc(sizeof(Node)*no_of_nodes, Managed);
#endif
                printf("alloc h_graph_mask\n");
#if adv2==0
	bool *h_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
#else
	bool *h_graph_mask = (bool*) xplacer_malloc(sizeof(bool)*no_of_nodes, Managed);
#endif
                printf("alloc h_updating_graph_mask\n");
#if adv3==0
	bool *h_updating_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
#else
	bool *h_updating_graph_mask = (bool*) xplacer_malloc(sizeof(bool)*no_of_nodes, Managed);
#endif
                printf("alloc h_graph_visited\n");
#if adv4==0
	bool *h_graph_visited = (bool*) malloc(sizeof(bool)*no_of_nodes);
#else
	bool *h_graph_visited = (bool*) xplacer_malloc(sizeof(bool)*no_of_nodes, Managed);
#endif

	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].starting = start;
		h_graph_nodes[i].no_of_edges = edgeno;
		h_graph_mask[i]=false;
		h_updating_graph_mask[i]=false;
		h_graph_visited[i]=false;
	}

	//read the source node from the file
	fscanf(fp,"%d",&source);
	source=0;

	//set the source node as true in the mask
	h_graph_mask[source]=true;
	h_graph_visited[source]=true;

	fscanf(fp,"%d",&edge_list_size);

	int id,cost;
                printf("alloc h_graph_edges\n");
#if adv5 == 0
	int* h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
#else
	int* h_graph_edges = (int*) xplacer_malloc(sizeof(int)*edge_list_size, Managed);
#endif
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i] = id;
	}

	if(fp)
		fclose(fp);    

	printf("Read File\n");

	//Copy the Node list to device memory
	Node* d_graph_nodes;
#if adv1 == 0
	d_graph_nodes = (Node*)xplacer_malloc(sizeof(Node)*no_of_nodes,GPU) ;
	xplacer_memcpy( (void**)&d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice,false) ;
#else
	xplacer_memcpy( (void**)&d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice,true) ;
#endif

	//Copy the Edge List to device Memory
	int* d_graph_edges;
#if adv5 == 0
	d_graph_edges = (int*)xplacer_malloc( sizeof(int)*edge_list_size,GPU) ;
	xplacer_memcpy( (void**)&d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice,false) ;
#else
	xplacer_memcpy( (void**)&d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice,true) ;
#endif

	//Copy the Mask to device memory
	bool* d_graph_mask;
#if adv2 == 0 
	d_graph_mask = (bool*)xplacer_malloc( sizeof(bool)*no_of_nodes,GPU) ;
	xplacer_memcpy( (void**)&d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice,false) ;
#else
	xplacer_memcpy( (void**)&d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice,true) ;
#endif

	bool* d_updating_graph_mask;
#if adv3 == 0 
	d_updating_graph_mask = (bool*)xplacer_malloc( sizeof(bool)*no_of_nodes,GPU) ;
	xplacer_memcpy( (void**)&d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice,false) ;
#else
	xplacer_memcpy( (void**)&d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice,true) ;
#endif

	//Copy the Visited nodes array to device memory
	bool* d_graph_visited;
#if adv4 == 0
	d_graph_visited = (bool*)xplacer_malloc( sizeof(bool)*no_of_nodes,GPU) ;
	xplacer_memcpy( (void**)&d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice,false) ;
#else
	xplacer_memcpy( (void**)&d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice,true) ;
#endif

	// allocate mem for the result on host side
#if adv6==0
	int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
#else
	int* h_cost = (int*) xplacer_malloc( sizeof(int)*no_of_nodes, Managed);
#endif
	for(int i=0;i<no_of_nodes;i++)
		h_cost[i]=-1;
	h_cost[source]=0;
	
	// allocate device memory for result
	int* d_cost;
#if adv6==0
	d_cost = (int*)xplacer_malloc( sizeof(int)*no_of_nodes,Managed);
	xplacer_memcpy( (void**)&d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice,false) ;
#else
	xplacer_memcpy( (void**)&d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice,true) ;
#endif

	//make a bool to check if the execution is over
	bool *d_over;
	hipMalloc( (void**) &d_over, sizeof(bool));

	printf("Copied Everything to GPU memory\n");

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	int k=0;
	printf("Start traversing the tree\n");

#if adv1==2
	hipMemAdvise(h_graph_nodes,sizeof(Node)*no_of_nodes, hipMemAdviseSetReadMostly, 0);
#elif adv1==3
	hipMemAdvise(h_graph_nodes,sizeof(Node)*no_of_nodes, hipMemAdviseSetPreferredLocation, 0);
#elif adv1==4
	hipMemAdvise(h_graph_nodes,sizeof(Node)*no_of_nodes, hipMemAdviseSetAccessedBy, 0);
#elif adv1==5
	hipMemAdvise(h_graph_nodes,sizeof(Node)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
#elif adv1==6
	hipMemAdvise(h_graph_nodes,sizeof(Node)*no_of_nodes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
#endif

#if adv2==2
	hipMemAdvise(h_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetReadMostly, 0);
#elif adv2==3
	hipMemAdvise(h_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, 0);
#elif adv2==4
	hipMemAdvise(h_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetAccessedBy, 0);
#elif adv2==5
	hipMemAdvise(h_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
#elif adv2==6
	hipMemAdvise(h_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
#endif

#if adv3==2
	hipMemAdvise(h_updating_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetReadMostly, 0);
#elif adv3==3
	hipMemAdvise(h_updating_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, 0);
#elif adv3==4
	hipMemAdvise(h_updating_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetAccessedBy, 0);
#elif adv3==5
	hipMemAdvise(h_updating_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
#elif adv3==6
	hipMemAdvise(h_updating_graph_mask,sizeof(bool)*no_of_nodes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
#endif

#if adv4==2
	hipMemAdvise(h_graph_visited,sizeof(bool)*no_of_nodes, hipMemAdviseSetReadMostly, 0);
#elif adv4==3
	hipMemAdvise(h_graph_visited,sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, 0);
#elif adv4==4
	hipMemAdvise(h_graph_visited,sizeof(bool)*no_of_nodes, hipMemAdviseSetAccessedBy, 0);
#elif adv4==5
	hipMemAdvise(h_graph_visited,sizeof(bool)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
#elif adv4==6
	hipMemAdvise(h_graph_visited,sizeof(bool)*no_of_nodes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
#endif

#if adv5==2
	hipMemAdvise(h_graph_edges,sizeof(int)*no_of_nodes, hipMemAdviseSetReadMostly, 0);
#elif adv5==3
	hipMemAdvise(h_graph_edges,sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, 0);
#elif adv5==4
	hipMemAdvise(h_graph_edges,sizeof(int)*no_of_nodes, hipMemAdviseSetAccessedBy, 0);
#elif adv5==5
	hipMemAdvise(h_graph_edges,sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
#elif adv5==6
	hipMemAdvise(h_graph_edges,sizeof(int)*no_of_nodes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
#endif

#if adv6==2
	hipMemAdvise(h_cost,sizeof(int)*no_of_nodes, hipMemAdviseSetReadMostly, 0);
#elif adv6==3
	hipMemAdvise(h_cost,sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, 0);
#elif adv6==4
	hipMemAdvise(h_cost,sizeof(int)*no_of_nodes, hipMemAdviseSetAccessedBy, 0);
#elif adv6==5
	hipMemAdvise(h_cost,sizeof(int)*no_of_nodes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
#elif adv6==6
	hipMemAdvise(h_cost,sizeof(int)*no_of_nodes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
#endif
//	setAdvise = hipMemAdviseSetPreferredLocation;

	bool stop;
	//Call the Kernel untill all the elements of Frontier are not false
	do
	{
		//if no thread changes this value then the loop stops
		stop=false;
		hipMemcpy( d_over, &stop, sizeof(bool), hipMemcpyHostToDevice) ;
		

		Kernel<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
		// check if kernel execution generated and error
		

		Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);
		// check if kernel execution generated and error
		
		hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
		k++;
  hipDeviceSynchronize();

	}
	while(stop);



	printf("Kernel Executed %d times\n",k);

	// copy result from device to host
//	xplacer_memcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost,true) ;

	//Store the result into a file
	FILE *fpo = fopen("result.txt","w");
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fpo,"%d) cost:%d\n",i,h_cost[i]);
	fclose(fpo);
	printf("Result stored in result.txt\n");


	// cleanup memory
#if adv1 == 0
	xplacer_free( h_graph_nodes, CPU);
	xplacer_free(d_graph_nodes,GPU);
#else
	xplacer_free( h_graph_nodes, Managed);
#endif
#if adv5 == 0
	xplacer_free( h_graph_edges, CPU);
	xplacer_free(d_graph_edges,GPU);
#else
	xplacer_free( h_graph_edges, Managed);
#endif
#if adv2 == 0
	xplacer_free( h_graph_mask, CPU);
	xplacer_free(d_graph_mask,GPU);
#else
	xplacer_free( h_graph_mask, Managed);
#endif
#if adv3 == 0
	xplacer_free( h_updating_graph_mask, CPU);
	xplacer_free(d_updating_graph_mask,GPU);
#else
	xplacer_free( h_updating_graph_mask, Managed);
#endif
#if adv4 == 0
	xplacer_free( h_graph_visited, CPU);
	xplacer_free(d_graph_visited,GPU);
#else
	xplacer_free( h_graph_visited, Managed);
#endif
#if adv6 == 0
	xplacer_free( h_cost, CPU);
	xplacer_free(d_cost,GPU);
#else
	xplacer_free( h_cost, Managed);
#endif
}
